#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include <math.h>

#define CHECK(call)                                                            \
  {                                                                            \
    const hipError_t error = call;                                            \
    if (error != hipSuccess) {                                                \
      printf("Error: %s:%d, ", __FILE__, __LINE__);                            \
      printf("code:%d, reason: %s\n", error, hipGetErrorString(error));       \
      exit(-10 * error);                                                       \
    }                                                                          \
  }

double cpuSecond() {
  struct timeval tp;
  gettimeofday(&tp, NULL);
  return ((double)tp.tv_sec + (double)tp.tv_usec * 1.e-6);
}

void initialData(float *ip, int size) {
  time_t t;
  srand((unsigned)time(&t));
  for (int i = 0; i < size; i++) {
    ip[i] = (float)(rand() & 0xFF) / 10.0f;
  }
}

void sumArraysOnHost(float *A, float *B, float *C, const int N) {
  for (int idx = 0; idx < N; idx++)
    C[idx] = A[idx] + B[idx];
}

__global__ void sumArraysOnGPU(float *A, float *B, float *C, int N) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N)
    C[i] = A[i] + B[i];
}

void checkResult(float *hostRef, float *gpuRef, const int N) {
  double epsilon = 1.0E-8;
  bool match = 1;
  for (int i = 0; i < N; i++) {
    if (fabsf(hostRef[i] - gpuRef[i]) > epsilon) {
      match = 0;
      printf("Arrays do not match!\n");
      printf("host %5.2f gpu %5.2f at current %d\n", hostRef[i], gpuRef[i], i);
      break;
    }
  }
  if (match)
    printf("Arrays match.\n\n");
}

int main(int argc, char **argv) {
  printf("%s Starting...\n", argv[0]);

  int dev = 0;
  hipDeviceProp_t deviceProp;
  CHECK(hipGetDeviceProperties(&deviceProp, dev));
  printf("Using Device %d: %s\n", dev, deviceProp.name);
  CHECK(hipSetDevice(dev));

  int nElem = 1 << 24;
  printf("Vector size %d\n", nElem);

  size_t nBytes = nElem * sizeof(float);
  float *h_A, *h_B, *hostRef, *gpuRef;
  h_A = (float *)malloc(nBytes);
  h_B = (float *)malloc(nBytes);
  hostRef = (float *)malloc(nBytes);
  gpuRef = (float *)malloc(nBytes);

  double iStart, iElaps;

  iStart = cpuSecond();
  initialData(h_A, nElem);
  initialData(h_B, nElem);
  iElaps = cpuSecond() - iStart;
  printf("initData Time elapsed %f sec\n", iElaps);

  memset(hostRef, 0, nBytes);
  memset(gpuRef, 0, nBytes);

  iStart = cpuSecond();
  sumArraysOnHost(h_A, h_B, hostRef, nElem);
  iElaps = cpuSecond() - iStart;
  printf("sumArraysOnHost Time elapsed %f sec\n", iElaps);

  float *d_A, *d_B, *d_C;
  CHECK(hipMalloc((float **)&d_A, nBytes));
  CHECK(hipMalloc((float **)&d_B, nBytes));
  CHECK(hipMalloc((float **)&d_C, nBytes));

  CHECK(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));
  CHECK(hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice));

  int iLen = 1024;
  dim3 block(iLen);
  dim3 grid((nElem + block.x - 1) / block.x);

  iStart = cpuSecond();
  sumArraysOnGPU<<<grid, block>>>(d_A, d_B, d_C, nElem);
  CHECK(hipDeviceSynchronize());
  iElaps = cpuSecond() - iStart;
  printf("sumArraysOnGPU <<<%d,%d>>> Time elapsed %f sec\n", grid.x, block.x, iElaps);

  CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));
  checkResult(hostRef, gpuRef, nElem);

  CHECK(hipFree(d_A));
  CHECK(hipFree(d_B));
  CHECK(hipFree(d_C));
  free(h_A);
  free(h_B);
  free(hostRef);
  free(gpuRef);

  return 0;
}

