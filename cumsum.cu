#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#define N 50


__global__ void add_vect(int *a, int *b, int *c){
    int idx = threadIdx.x;
    c[idx] = a[idx] + b[idx];
}

int main(){
    int *a, *b, *c;
    int *dev_a, *dev_b, *dev_c;
    int nBytes = N * sizeof(int);

    a = (int *) malloc(nBytes);
    for(int i = 0; i < N; i++){
        a[i] = rand();
        printf("%d\n", a[i]);
    }
    b = (int *) malloc(nBytes);
    c = (int *) malloc(nBytes);

    hipMalloc((void**) &dev_a, nBytes);
    hipMalloc((void**) &dev_b, nBytes);
    hipMalloc((void**) &dev_c, nBytes);

    hipMemcpy(dev_a, a, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, nBytes, hipMemcpyHostToDevice);

    add_vect<<<1, N>>>(dev_a, dev_b, dev_c);

    hipMemcpy(c, dev_c, nBytes, hipMemcpyDeviceToHost);

    printf("%d\n", c[3]);

    free(a);
    free(b);
    free(c);

    hipFree(a);
    hipFree(b);
    hipFree(c);
}

main()
